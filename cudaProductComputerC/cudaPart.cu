#include "hip/hip_runtime.h"
#pragma once

#include "AlgebraSetting.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void simpleCopyKernel(int* in_pos, int* out_pos) {
	int tx = threadIdx.x;
	out_pos[tx] = in_pos[tx]*2;
}

extern "C" void 
cudaCalculateProducts(int summandCountPMTransformedZI, int* positionsPMTransformedZI, int* lengthsPMTransformedZI, float* coefficentsPMTransformedZI) {
// allocate memory pmTransformedZI
	float* cin_coefficentsPMTransformedZI;
	int size = summandCountPMTransformedZI * sizeof(float);
	assert(hipMalloc((void**) &cin_coefficentsPMTransformedZI, size) == hipSuccess);
	assert(hipMemcpy(cin_coefficentsPMTransformedZI, coefficentsPMTransformedZI, size, hipMemcpyHostToDevice) == hipSuccess);

	int* cin_positionsPMTransformedZI;
	size = BLADECOUNT*sizeof(int);
	assert(hipMalloc((void**) &cin_positionsPMTransformedZI, size) == hipSuccess);
	assert(hipMemcpy(cin_positionsPMTransformedZI, positionsPMTransformedZI, size, hipMemcpyHostToDevice) == hipSuccess);

	int* cin_lengthsPMTransformedZI;
	assert(hipMalloc((void**) &cin_lengthsPMTransformedZI, size) == hipSuccess);
	assert(hipMemcpy(cin_lengthsPMTransformedZI, lengthsPMTransformedZI, size, hipMemcpyHostToDevice) == hipSuccess);

	// allocate memory for output
	//TODO first some testing
	int* cout_out;
	assert(hipMalloc((void**) &cout_out, BLADECOUNT*sizeof(int)) == hipSuccess);
	assert(hipMemset(cout_out,0,BLADECOUNT*sizeof(int)) == hipSuccess);

	// TODO CPU: collect results on host-memory, print it, or store it into a binary file for loading in Gaalop
	dim3 dimBlock(32,32,1);
	dim3 dimGrid(BLADECOUNT/32,BLADECOUNT/32,1); //BLADECOUNT<=11 muss gelten (f�r diese Hardware)
	std::cout << "Number of blocks: " << BLADECOUNT/32 << " x " << BLADECOUNT/32 << std::endl;

	simpleCopyKernel<<<dimGrid,dimBlock>>>(cin_positionsPMTransformedZI, cout_out);


	// retrieve data from gpu global memory
	int out[BLADECOUNT];

	assert(hipMemcpy(out, cout_out, BLADECOUNT*sizeof(int), hipMemcpyDeviceToHost) == hipSuccess);

	//for (int i = 0;i<BLADECOUNT;i++) 
	//	std::cout << out[i] << std::endl;

	// Free memory
	assert(hipFree(cout_out) == hipSuccess);
	assert(hipFree(cin_coefficentsPMTransformedZI) == hipSuccess);
	assert(hipFree(cin_positionsPMTransformedZI) == hipSuccess);
	assert(hipFree(cin_lengthsPMTransformedZI) == hipSuccess);
}