#include "hip/hip_runtime.h"
#pragma once

#include "AlgebraSetting.h"
#include <assert.h>

#include <iostream>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#define THREAD_DIM_X 16
#define THREAD_DIM_Y 16

/**
	Returns the number of setted bits in a number
	@returns The number of setted bits
**/
__device__
int count(int bits) {
	if (bits == 0) return 0;
	int count = 0;
	int mask = 1;
	for (int bit = 0; bit < MAXBITCOUNT; ++bit) {
		if ((bits & mask) > 0) count++;
		mask <<= 1;
	}
	return count;
}

// Computes �reordering sign� to get into canonical order.
// Arguments 'a' and 'b' are both bitmaps representing basis blades.
// Copy a!
// This method is taken from the dissertation of Daniel Fontijne - Efficient Implementation of Gemoetric Algebra
__device__
	int canonicalReorderingSign(int blade1, int blade2) {
	// Count the number of basis vector swaps required to
	// get 'a' and 'b' into canonical order.
	blade1 >>= 1;
	int sum = 0;
	while (blade1 != 0)
	{
		// the function bitCount() counts the number of
		// 1-bits in the argument
		int aAndB = blade1 & blade2;
		sum += count(aAndB);
		blade1 >>= 1;
	}
	// even number of swaps -> return 1
	// odd number of swaps -> return -1
	return ((sum & 1) == 0) ? 1 : -1;
}

__device__ void outerKernelCalc(
	int* c_positionsPMTransformedZI, int* c_lengthsPMTransformedZI, float* c_coefficentsPMTransformedZI, int* c_bladesPMTransformedZI,
	int* c_positionsZITransformedPM, int* c_lengthsZITransformedPM, float* c_coefficentsZITransformedPM, int* c_bladesZITransformedPM,
	float* accumulatorCoefficients, int x, int y, int idInBlock
	) {
	
	
	int posX = c_positionsPMTransformedZI[x];
	int posY = c_positionsPMTransformedZI[y];
	int lenX = c_lengthsPMTransformedZI[x];
	int lenY = c_lengthsPMTransformedZI[y];

	for (int blade1Id = posX;blade1Id < posX+lenX; blade1Id++) 
		for (int blade2Id = posY;blade2Id < posY+lenY; blade2Id++) {
			float coeff1 = c_coefficentsPMTransformedZI[blade1Id];
			int blade1 = c_bladesPMTransformedZI[blade1Id];
			float coeff2 = c_coefficentsPMTransformedZI[blade2Id];
			int blade2 = c_bladesPMTransformedZI[blade2Id];

			float coeff;
			int result;

			//outer product
			if ((blade1 & blade2) == 0) {
				//outer product is not null
				result = blade1 ^ blade2; // also "or"-operation is possible
				coeff = coeff1*coeff2*canonicalReorderingSign(blade1, blade2);

				int pos = c_positionsZITransformedPM[result];
				int len = c_lengthsZITransformedPM[result];
				for (int i=pos;i<pos+len;i++) {
					float coeffA = c_coefficentsZITransformedPM[i];
					int bladeA = c_bladesZITransformedPM[i];

					accumulatorCoefficients[bladeA+idInBlock*BLADECOUNT] += coeff*coeffA;
				}

			} else {
				//outer product is null
				coeff = 0;
				result = 0;
			}
			
		}

}
	
 __global__ void outerKernelPre( //TODO wrong implementation, count is always 32 for dim=5, debug this by copying into normal function
	int* c_positionsPMTransformedZI, int* c_lengthsPMTransformedZI, float* c_coefficentsPMTransformedZI, int* c_bladesPMTransformedZI,
	int* c_positionsZITransformedPM, int* c_lengthsZITransformedPM, float* c_coefficentsZITransformedPM, int* c_bladesZITransformedPM,
	int* c_outCounts
	) {
	__shared__ float accumulatorCoefficients[BLADECOUNT*THREAD_DIM_X*THREAD_DIM_Y];
	for (int i=0;i<BLADECOUNT*THREAD_DIM_X*THREAD_DIM_Y;i++)
		accumulatorCoefficients[i] = 0;

	int x = threadIdx.x+THREAD_DIM_X*blockIdx.x;
	int y = threadIdx.y+THREAD_DIM_Y*blockIdx.y;
	int idInBlock = threadIdx.x*THREAD_DIM_X+threadIdx.y;

	__syncthreads();

	outerKernelCalc(
		c_positionsPMTransformedZI, c_lengthsPMTransformedZI, c_coefficentsPMTransformedZI, c_bladesPMTransformedZI,
	    c_positionsZITransformedPM, c_lengthsZITransformedPM, c_coefficentsZITransformedPM, c_bladesZITransformedPM,
		accumulatorCoefficients, x, y, idInBlock
		);

	// zip accumulator
	int index = 0;
	for (int i=0;i<BLADECOUNT;i++) 
		if (abs(accumulatorCoefficients[i+idInBlock*BLADECOUNT]) > 10E-4)  
			index++;

	c_outCounts[x*BLADECOUNT+y] = index;
}



extern "C" void 
cudaCalculateProducts(
	int summandCountPMTransformedZI, int* positionsPMTransformedZI, 
	int* lengthsPMTransformedZI, float* coefficentsPMTransformedZI, int* bladesPMTransformedZI,
	int summandCountZITransformedPM, int* positionsZITransformedPM, 
	int* lengthsZITransformedPM, float* coefficentsZITransformedPM, int* bladesZITransformedPM
	) {
		
// allocate memory pmTransformedZI
	float* cin_coefficentsPMTransformedZI;
	int size = summandCountPMTransformedZI * sizeof(float);
	checkCudaErrors(hipMalloc((void**) &cin_coefficentsPMTransformedZI, size));
	checkCudaErrors(hipMemcpy(cin_coefficentsPMTransformedZI, coefficentsPMTransformedZI, size, hipMemcpyHostToDevice));

	int* cin_bladesPMTransformedZI;
	size = summandCountPMTransformedZI * sizeof(int);
	checkCudaErrors(hipMalloc((void**) &cin_bladesPMTransformedZI, size));
	checkCudaErrors(hipMemcpy(cin_bladesPMTransformedZI, bladesPMTransformedZI, size, hipMemcpyHostToDevice));

	int* cin_positionsPMTransformedZI;
	size = BLADECOUNT*sizeof(int);
	checkCudaErrors(hipMalloc((void**) &cin_positionsPMTransformedZI, size));
	checkCudaErrors(hipMemcpy(cin_positionsPMTransformedZI, positionsPMTransformedZI, size, hipMemcpyHostToDevice));

	int* cin_lengthsPMTransformedZI;
	checkCudaErrors(hipMalloc((void**) &cin_lengthsPMTransformedZI, size));
	checkCudaErrors(hipMemcpy(cin_lengthsPMTransformedZI, lengthsPMTransformedZI, size, hipMemcpyHostToDevice));
	
// allocate memory ziTransformedPM
	float* cin_coefficentsZITransformedPM;
	size = summandCountZITransformedPM * sizeof(float);
	checkCudaErrors(hipMalloc((void**) &cin_coefficentsZITransformedPM, size));
	checkCudaErrors(hipMemcpy(cin_coefficentsZITransformedPM, coefficentsZITransformedPM, size, hipMemcpyHostToDevice));

	int* cin_bladesZITransformedPM;
	size = summandCountZITransformedPM * sizeof(int);
	checkCudaErrors(hipMalloc((void**) &cin_bladesZITransformedPM, size));
	checkCudaErrors(hipMemcpy(cin_bladesZITransformedPM, bladesZITransformedPM, size, hipMemcpyHostToDevice));

	int* cin_positionsZITransformedPM;
	size = BLADECOUNT*sizeof(int);
	checkCudaErrors(hipMalloc((void**) &cin_positionsZITransformedPM, size));
	checkCudaErrors(hipMemcpy(cin_positionsZITransformedPM, positionsZITransformedPM, size, hipMemcpyHostToDevice));

	int* cin_lengthsZITransformedPM;
	checkCudaErrors(hipMalloc((void**) &cin_lengthsZITransformedPM, size));
	checkCudaErrors(hipMemcpy(cin_lengthsZITransformedPM, lengthsZITransformedPM, size, hipMemcpyHostToDevice));
	
// allocate memory for output
	int* c_outCounts;
	checkCudaErrors(hipMalloc((void**) &c_outCounts, BLADECOUNT*BLADECOUNT*sizeof(int)));
	checkCudaErrors(hipMemset(c_outCounts,0,BLADECOUNT*BLADECOUNT*sizeof(int)));
	
	// TODO CPU: collect results on host-memory, print it, or store it into a binary file for loading in Gaalop
	dim3 dimBlock(THREAD_DIM_X,THREAD_DIM_Y,1);
	dim3 dimGrid(BLADECOUNT/THREAD_DIM_X,BLADECOUNT/THREAD_DIM_Y,1); //BLADECOUNT<=11 muss gelten (f�r diese Hardware)
	std::cout << "Number of blocks: " << BLADECOUNT/THREAD_DIM_X << " x " << BLADECOUNT/THREAD_DIM_Y << std::endl;
	//TODO inner and geo kernel
	
	
	//precompute
	outerKernelPre<<<dimGrid,dimBlock>>>(
		cin_positionsPMTransformedZI, cin_lengthsPMTransformedZI, cin_coefficentsPMTransformedZI, cin_bladesPMTransformedZI,
		cin_positionsZITransformedPM, cin_lengthsZITransformedPM, cin_coefficentsZITransformedPM, cin_bladesZITransformedPM,
		c_outCounts
		);
	
	int* out = new int[BLADECOUNT*BLADECOUNT];

	checkCudaErrors(hipMemcpy(out, c_outCounts, BLADECOUNT*BLADECOUNT*sizeof(int), hipMemcpyDeviceToHost));
	

	int max = 0;
	int sum = 0;
	for (int i=0;i<BLADECOUNT*BLADECOUNT;i++) {
		if (max < out[i]) 
			max = out[i];
		sum += out[i];
		//std::cout << i << ": " << out[i] << std::endl;
	}

	std::cout << "Max: " << max << " , Sum: " << sum << std::endl;
	getchar();
	// retrieve data from gpu global memory

	// Free memory
	
	checkCudaErrors(hipFree(c_outCounts));
	checkCudaErrors(hipFree(cin_bladesZITransformedPM));
	checkCudaErrors(hipFree(cin_coefficentsZITransformedPM));
	checkCudaErrors(hipFree(cin_positionsZITransformedPM));
	checkCudaErrors(hipFree(cin_lengthsZITransformedPM));
	
	checkCudaErrors(hipFree(cin_bladesPMTransformedZI));
	checkCudaErrors(hipFree(cin_coefficentsPMTransformedZI));
	checkCudaErrors(hipFree(cin_positionsPMTransformedZI));
	checkCudaErrors(hipFree(cin_lengthsPMTransformedZI));
	delete[] out;
	out = 0;
	hipDeviceReset();
}