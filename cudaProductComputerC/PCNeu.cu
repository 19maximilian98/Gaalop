#include "hip/hip_runtime.h"
// PCNeu.cpp : Definiert den Einstiegspunkt f�r die Konsolenanwendung.
//

#include "Definitions.h"
#include "SignedBlade.h"
#include "GAMethods.h"
#include <iostream>

#include "BladelistCreator.h"

#include "InnerProductComputing.h"
#include "OuterGeoProductComputing.h"
#include "Grouper.h"
#include "Outputter.h"

#include <boost/unordered_map.hpp>

#include <time.h>
#include <fstream>

#include "BitWriter.h"
#include "BitReader.h"


#define COMPUTE_INNER_PRODUCT
#define COMPUTE_OUTER_AND_GEO_PRODUCT

#define PRINT_TO_FILE

void printBladelist(Bladelist& list, void (*printer) (Blade&, std::ostream&)) {
	std::fstream out("D:\\blades.csv",std::fstream::out);
	int index = 0;
	for (Bladelist::iterator sblade1=list.begin(); sblade1 != list.end(); ++sblade1) {
		SumOfBlades&s = *sblade1;
		out << index << ": ";
		output(s,printer,out);
		out << std::endl;
		index++;
	}
	out.close();
}

#include "CalcThread.h"

#define BLADECOUNT 32

// CUDA runtime
#include <hip/hip_runtime.h>
#include <assert.h>
#include <iostream>

#include "Definitions.h"

void cudaCalculateProducts(SumOfBlades* pmTransformedZI);

__global__ void simpleCopyKernel(int* in_pos, int* out_pos) {
	int tx = threadIdx.x;
	out_pos[tx] = in_pos[tx]*2;
}

void cudaCalculateProducts(SumOfBlades* pmTransformedZI) {
	// == allocate memories ==
	
	// retrieve informations about positions, lenghts and number of elements
	int positionsPMTransformedZI[BLADECOUNT];
	int lengthsPMTransformedZI[BLADECOUNT];
	int summandCountPMTransformedZI = 0;
	int position = 0;
	for (int i=0;i<BLADECOUNT;i++) {
		positionsPMTransformedZI[i] = position;
		lengthsPMTransformedZI[i] = pmTransformedZI[i].size();
		summandCountPMTransformedZI += lengthsPMTransformedZI[i];
		position += lengthsPMTransformedZI[i];
	}

	float* coefficentsPMTransformedZI = new float[summandCountPMTransformedZI];
	int index = 0;
	for (int i=0;i<BLADECOUNT;i++) 
		for (SumOfBlades::iterator sblade1 = pmTransformedZI[i].begin(); sblade1 != pmTransformedZI[i].end(); ++sblade1) {
			coefficentsPMTransformedZI[index] = (*sblade1).coefficient;
			index++;
		}

	// allocate memory pmTransformedZI
	float* cin_coefficentsPMTransformedZI;
	int size = summandCountPMTransformedZI * sizeof(float);
	assert(hipMalloc((void**) &cin_coefficentsPMTransformedZI, size) == hipSuccess);
	assert(hipMemcpy(cin_coefficentsPMTransformedZI, coefficentsPMTransformedZI, size, hipMemcpyHostToDevice) == hipSuccess);

	int* cin_positionsPMTransformedZI;
	size = BLADECOUNT*sizeof(int);
	assert(hipMalloc((void**) &cin_positionsPMTransformedZI, size) == hipSuccess);
	assert(hipMemcpy(cin_positionsPMTransformedZI, positionsPMTransformedZI, size, hipMemcpyHostToDevice) == hipSuccess);

	int* cin_lengthsPMTransformedZI;
	assert(hipMalloc((void**) &cin_lengthsPMTransformedZI, size) == hipSuccess);
	assert(hipMemcpy(cin_lengthsPMTransformedZI, lengthsPMTransformedZI, size, hipMemcpyHostToDevice) == hipSuccess);

	// allocate memory for output
	//TODO first some testing
	int* cout_out;
	assert(hipMalloc((void**) &cout_out, BLADECOUNT*sizeof(int)) == hipSuccess);
	assert(hipMemset(cout_out,0,BLADECOUNT*sizeof(int)) == hipSuccess);

	// TODO CPU: collect results on host-memory, print it, or store it into a binary file for loading in Gaalop
	dim3 dimBlock(1,1,1);
	dim3 dimGrid(BLADECOUNT,1,1);
	simpleCopyKernel<<<dimGrid,dimBlock>>>(cin_positionsPMTransformedZI, cout_out);


	// retrieve data from gpu global memory
	int out[BLADECOUNT];

	assert(hipMemcpy(out, cout_out, BLADECOUNT*sizeof(int), hipMemcpyDeviceToHost) == hipSuccess);

	for (int i = 0;i<BLADECOUNT;i++) 
		std::cout << out[i] << std::endl;

	// Free memory
	assert(hipFree(cout_out) == hipSuccess);
	assert(hipFree(cin_coefficentsPMTransformedZI) == hipSuccess);
	assert(hipFree(cin_positionsPMTransformedZI) == hipSuccess);
	assert(hipFree(cin_lengthsPMTransformedZI) == hipSuccess);
}





/**
	Creates the producttables of a geometric algebra using CUDA.
 **/
int main(int argc, char* argv[])
{
	time_t start;
	time(&start);

	// CPU: create bladeListZI
	Bladelist bladelistZI;
	initializeBladelist(bladelistZI);
	
	// CPU: transform bladeListZI into Sum-Of-PMBlades variable pmTransformedZI
	SumOfBlades pmTransformedZI[BLADECOUNT];
	for (int index=0;index<BLADECOUNT;index++) {
		SumOfBlades b;
		b.push_back(SignedBlade(Blade(index)));
		SumOfBlades& pm = pmTransformedZI[index];
		basetransformationZeroInfToPlusMinus(b, pm);
		group(pm);
	}
	
	// CPU: create transformations from PM to ZI 
	SumOfBlades ziTransformedPM[BLADECOUNT];
	for (int index=0;index<BLADECOUNT;index++) {
		SumOfBlades b;
		b.push_back(SignedBlade(Blade(index)));
		SumOfBlades& zi = ziTransformedPM[index];
		basetransformationPlusMinusToZeroInf(b,zi);
		group(zi);
	}

	// TODO CUDA: calculate all products in PM, transform it into ZI, store it on host-memory
	cudaCalculateProducts(pmTransformedZI);
	

	time_t ende;
	time(&ende);
	std::cout << "Ready in " << difftime(ende, start) << " seconds" << std::endl;
	getchar();
	return 0;
}


/*
//int maxNumber = 0; //TODO
	//std::cout << "MaxNumber = " << maxNumber <<std::endl;
// komprimieren
	int number = 2;
    int bitCount2 = 1;
    while (number < maxNumber+1) {
        bitCount2++;
        number *= 2;
    }
	*/